
#include <hip/hip_runtime.h>


__global__ void kernel_init(float* matrix, int ld, int rows, int cols, float value) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols) {
        matrix[row+col*ld] = value;
    }
}

__global__ void kernel_addValue(float* matrix, int ld, float* output, int output_ld, int rows, int cols, float value) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols) {
        output[row+col*output_ld] = matrix[row+col*ld] + value;
    }
}
__global__ void kernel_scale(float* matrix, int ld, float* output, int output_ld, int rows, int cols, float value) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols) {
        output[row+col*output_ld] = matrix[row+col*ld] * value;
    }
}
__global__ void kernel_add(float* left_op, int left_op_ld, float* right_op, int right_op_ld, float* output, int output_ld, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols) {
        output[row+col*output_ld] = left_op[row+col*left_op_ld] + right_op[row+col*right_op_ld];
    }
}

__global__ void kernel_aYpb(float a, float b, float* Y, int Y_ld, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols) {
        Y[row+col*Y_ld] = a*Y[row+col*Y_ld] + b;
    }
}

__global__ void kernel_convolution(float* input, int inputRows, int inputCols, int inputLd,
                            float* kernel, int kernelRows, int kernelCols, int kernelLd,
                            int rowStep, int colStep, float* output, int outputLd) {

    int row = (blockIdx.y * blockDim.y + threadIdx.y) * rowStep;
    int col = (blockIdx.x * blockDim.x + threadIdx.x) * colStep;

    if (row <= inputRows - kernelRows && col <= inputCols - kernelCols) {
        int i, j;
        output[row+col*inputLd] = 0;
        for (i=0; i<kernelRows; i++) {
            for (j=0; j<kernelCols; j++) {
                output[row+col*outputLd] += kernel[i+j*kernelLd] * input[(row+i)+(col+j)*inputLd];
            }
        }
    }

}

extern "C" {
    void MatrixKernel_init(float* matrix, int ld, int rows, int cols, float value, hipStream_t stream) {
        dim3 blockDim(32, 32);
        dim3 gridDim((rows + blockDim.x - 1) / blockDim.x, (cols + blockDim.y - 1) / blockDim.y);
        kernel_init <<<gridDim, blockDim, 0, stream>>> (matrix, ld, rows, cols, value);
    }

    void MatrixKernel_addValue(float* matrix, int ld, float* output, int output_ld, int rows, int cols, float value, hipStream_t stream) {
        dim3 blockDim(32, 32);
        dim3 gridDim((rows + blockDim.x - 1) / blockDim.x, (cols + blockDim.y - 1) / blockDim.y);
        kernel_addValue <<<gridDim, blockDim, 0, stream>>> (matrix, ld, output, output_ld, rows, cols, value);
    }
    void MatrixKernel_scale(float* matrix, int ld, float* output, int output_ld, int rows, int cols, float value, hipStream_t stream) {
        dim3 blockDim(32, 32);
        dim3 gridDim((rows + blockDim.x - 1) / blockDim.x, (cols + blockDim.y - 1) / blockDim.y);
        kernel_scale <<<gridDim, blockDim, 0, stream>>> (matrix, ld, output, output_ld, rows, cols, value);
    }
    void MatrixKernel_add(float* left_op, int left_op_ld, float* right_op, int right_op_ld, float* output, int output_ld, int rows, int cols, hipStream_t stream) {
        dim3 blockDim(32, 32);
        dim3 gridDim((rows + blockDim.x - 1) / blockDim.x, (cols + blockDim.y - 1) / blockDim.y);
        kernel_add <<<gridDim, blockDim, 0, stream>>> (left_op, left_op_ld, right_op, right_op_ld, output, output_ld, rows, cols);
    }

    void MatrixKernel_aYpb(float a, float b, float* Y, int Y_ld, int rows, int cols, hipStream_t stream) {
        dim3 blockDim(32, 32);
        dim3 gridDim((rows + blockDim.x - 1) / blockDim.x, (cols + blockDim.y - 1) / blockDim.y);
        kernel_aYpb <<<gridDim, blockDim, 0, stream>>> (a, b, Y, Y_ld, rows, cols);
    }

    void MatrixKernel_convolution(float* input, int inputRows, int inputCols, int inputLd,
                                float* kernel, int kernelRows, int kernelCols, int kernelLd,
                                int rowStep, int colStep, float* output, int outputLd, hipStream_t stream) {

        dim3 blockDim(32, 32);
        dim3 gridDim((inputRows + blockDim.x - 1) / blockDim.x, (inputCols + blockDim.y - 1) / blockDim.y);
        kernel_convolution <<<gridDim, blockDim, 0, stream>>> (input, inputRows, inputCols, inputLd,
                                                        kernel, kernelRows, kernelCols, kernelLd,
                                                        rowStep, colStep, output, outputLd);

    }
}