
#include <hip/hip_runtime.h>


__global__ void convolution(float* input, int inputRows, int inputCols, int inputLd,
                            float* kernel, int kernelRows, int kernelCols, int kernelLd,
                            int rowStep, int colStep, float* output, int outputLd) {

    int row = (blockIdx.y * blockDim.y + threadIdx.y) * rowStep;
    int col = (blockIdx.x * blockDim.x + threadIdx.x) * colStep;

    if (row <= inputRows - kernelRows && col <= inputCols - kernelCols) {
        int i, j;
        output[row+col*outputLd] = 0;
        for (i=0; i<kernelRows; i++) {
            for (j=0; j<kernelCols; j++) {
                output[row+col*outputLd] += kernel[i+j*kernelLd] * input[(row+i)+(col+j)*inputLd];
            }
        }
    }

}

extern "C" {
    void Matrix_convolution(float* input, int inputRows, int inputCols, int inputLd,
                                float* kernel, int kernelRows, int kernelCols, int kernelLd,
                                int rowStep, int colStep, float* output, int outputLd, hipStream_t stream) {

        dim3 blockDim(32, 32);
        dim3 gridDim((inputRows + blockDim.x - 1) / blockDim.x, (inputCols + blockDim.y - 1) / blockDim.y);
        convolution <<<gridDim, blockDim, 0, stream>>> (input, inputRows, inputCols, inputLd,
                                                        kernel, kernelRows, kernelCols, kernelLd,
                                                        rowStep, colStep, output, outputLd);
    }
}